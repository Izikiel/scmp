#include "utils.h"

void __cudaCheck(hipError_t err, const char* file, const int line)
{
    if ( hipSuccess != err) {
        fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",
                file, line, (int)err, hipGetErrorString( err ) );
        exit(-1);
    }
}