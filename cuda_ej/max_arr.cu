#include "hip/hip_runtime.h"
#include "stdio.h"
#include "hip/hip_runtime.h"

__global__ void max_arr(int *arr, int N, int *max)
{
    __shared__ int iter = 1;

    for (int i = N; i > 0; i >>= iter, iter++) {
        __syncthreads();

    }

}

int main(int argc, char const *argv[])
{
    int N = 10;
    int arr[] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10};
    int max;
    int *d_arr;
    int *d_max;
    hipMalloc((void **)&d_arr, N * sizeof(int));
    hipMalloc((void **)&d_max, sizeof(int));

    hipMemcpy(d_arr, arr, N * sizeof(int), hipMemcpyHostToDevice);
    max_arr <<< 1, 10>>>(d_arr, N, d_max);
    hipMemcpy(arr, d_arr, N * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&max, d_max, sizeof(int), hipMemcpyDeviceToHost);

    printf("Maximo del array %d\n", max);

    hipFree(d_arr);
    hipFree(d_max);

    return 0;
}