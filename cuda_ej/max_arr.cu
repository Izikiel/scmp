#include "hip/hip_runtime.h"
#include "utils.h"

__global__ void max_arr(int* arr, int N, int* max)
{
    __shared__ int iter = 1;

    for (int i = N; i > 0; i >>= iter, iter++) {
        __syncthreads();

    }

}

int main(int argc, char const* argv[])
{
    int N = 10;
    int arr[] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10};
    int max;
    int* d_arr;
    int* d_max;
    cudaCheck(
        hipMalloc((void**)&d_arr, N * sizeof(int))
    );
    cudaCheck(
        hipMalloc((void**)&d_max, sizeof(int))
    );

    cudaCheck(
        hipMemcpy(d_arr, arr, N * sizeof(int), hipMemcpyHostToDevice)
    );

    max_arr <<< 1, 10>>>(d_arr, N, d_max);

    cudaCheck(
        hipMemcpy(arr, d_arr, N * sizeof(int), hipMemcpyDeviceToHost)
    );
    cudaCheck(
        hipMemcpy(&max, d_max, sizeof(int), hipMemcpyDeviceToHost)
    );

    printf("Maximo del array %d\n", max);

    cudaCheck(
        hipFree(d_arr)
    );
    cudaCheck(
        hipFree(d_max)
    );

    return 0;
}